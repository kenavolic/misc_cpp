#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <hipblas.h>

#include <iostream>
#include <cmath>
#include <algorithm>

#if __CUDA_ARCH__ == 500

#warning "__CUDA_ARCH__ 500"

#endif

// - https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html
// - https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
// - https://docs.nvidia.com/cuda/cuda-compiler-driver-nvcc/index.html
// - https://docs.nvidia.com/cuda/cuda-runtime-api/ 


// __global__ Define a CUDA kernel function must return void
// called from host code and executed on device
// __device_ Called and executed from device
//
// The kernel function is executed by N threads in parallel
//
__global__
void add(uint64_t size, float* arr1, float* arr2) { 
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;

  if (row < size && col < size) {
    arr1[row*size + col] = arr1[row*size + col] + arr2[row*size + col];
  }
}

__global__
void mul(uint64_t size, float* arr1, float* arr2, float* out) {
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;

  if (row < size && col < size) {
    float res{};
    for (uint64_t s = 0; s < size; s++) {
      res += arr1[row*size + s] * arr2[s*size + col];
    }
    out[row*size+col] = res;
  }
}

__global__
void mul_tile(uint64_t size, float* arr1, float* arr2, float* out) {
  int realRow = threadIdx.y + blockIdx.y * blockDim.y; 
  int realCol = threadIdx.x + blockIdx.x * blockDim.x;

  if (realRow > size || realCol > size) {
    return;
  }

  int row = threadIdx.y;
  int col = threadIdx.x;

  float Cvalue = 0;
  // for each sublocks, load Asubi and Bsubi to shared memory
  // compute multiplication
  for (uint64_t sub = 0; sub < size / 16U; sub++) {
    // load Asubi et bsubi
    __shared__ float Asubi[16][16];
    __shared__ float Bsubi[16][16];
    Asubi[row][col] = arr1[realRow*size+col+sub*16];
    Bsubi[row][col] = arr2[(row+sub*16)*size+realCol];
    __syncthreads();
    for (uint64_t e = 0; e < 16; e++) {
      Cvalue += Asubi[row][e] * Bsubi[e][col];
    }
    __syncthreads();
  }
  out[realRow*size+realCol] = Cvalue;
}

void mul_blas(const int size, const float *A, const float *B, float *C) {
     int lda=size,ldb=size,ldc=size;
     const float alf = 1;
     const float bet = 0;
     const float *alpha = &alf;
     const float *beta = &bet;
 
     // Create a handle for CUBLAS
     hipblasHandle_t handle;
     auto res = hipblasCreate(&handle);

     if (res != HIPBLAS_STATUS_SUCCESS) {
      std::cout << "cublas handle error " << res << std::endl;
     }
 
     // Do the actual multiplication
     // https://stackoverflow.com/questions/56043539/cublassgemm-row-major-multiplication
     // for the row major operation
     hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size, size, alpha, B, lda, A, ldb, beta, C, ldc);
 
     // Destroy the handle
     hipblasDestroy(handle);
}

void compute(float*a, float*b, float*c, size_t count, bool useLib = false) {
  const uint64_t kCount = count;

  /////////////////////////////////////////////////////////////////////////////
  /////////////////////////////// Initialization
  /////////////////////////////////////////////////////////////////////////////

  // check config before
  // hipGetDeviceCount
  // SM Version
  int devicesCount{-666};
  auto cudaStatus = hipGetDeviceCount(&devicesCount);
  
  if (cudaStatus != hipSuccess) {
      std::cerr << "Failed to get device count with error " << static_cast<int>(cudaStatus) << std::endl;
      return;
  }

  std::cout << "device count: " << devicesCount << std::endl;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  // 6.1 on my quadro = Pascal
  std::cout << "compute cap: " << prop.major << "." << prop.minor << std::endl;
  //std::cout << "the cuda api version: " << CUDA_VERSION << std::endl;

  // check the concurrent kernel prop
  std::cout << "concurrent kernel: " << prop.concurrentKernels << std::endl;

  /////////////////////////////////////////////////////////////////////////////
  /////////////////////////////// Create device container
  /////////////////////////////////////////////////////////////////////////////

  std::cout << "start computation" << std::endl;

  float*arr1{nullptr};
  float*arr2{nullptr};
  float*mulResult{nullptr};
  cudaStatus = hipMallocManaged(&arr1, kCount*kCount*sizeof(float));

  if (cudaStatus != hipSuccess) {
      std::cerr << "Failed to allocated first memory with error " << static_cast<int>(cudaStatus) << std::endl;
      return;
  }

  cudaStatus = hipMallocManaged(&arr2, kCount*kCount*sizeof(float));

  if (cudaStatus != hipSuccess) {
      std::cerr << "Failed to allocated second memory with error " << static_cast<int>(cudaStatus) << std::endl;
      return;
  }

  cudaStatus = hipMallocManaged(&mulResult, kCount*kCount*sizeof(float));

  if (cudaStatus != hipSuccess) {
      std::cerr << "Failed to allocated third memory with error " << static_cast<int>(cudaStatus) << std::endl;
      return;
  }

  std::cout << "memory allocated" << std::endl;

  hipMemcpy(arr1, a, sizeof(float) * kCount * kCount, hipMemcpyHostToDevice);
  hipMemcpy(arr2, b, sizeof(float) * kCount * kCount, hipMemcpyHostToDevice);

  std::cout << "array initialized" << std::endl;

  /////////////////////////////////////////////////////////////////////////////
  /////////////////////////////// Perform arithmetic operation
  /////////////////////////////////////////////////////////////////////////////

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks(kCount / threadsPerBlock.x, kCount / threadsPerBlock.y);

  if (!useLib) {
    std::cout << "using raw cuda" << std::endl;
    mul_tile<<<numBlocks, threadsPerBlock>>>(kCount, arr1, arr2, mulResult);
  } else {
    std::cout << "using cublas lib" << std::endl;
    mul_blas(kCount, arr1, arr2, mulResult);
  }
  
  hipDeviceSynchronize();
  add<<<numBlocks, threadsPerBlock>>>(kCount, mulResult, mulResult);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // transfer the result in c
  hipMemcpy(c, mulResult, sizeof(float) * kCount * kCount, hipMemcpyDeviceToHost);

  hipFree(arr1);
  hipFree(arr2);
  hipFree(mulResult);

  hipProfilerStop();
}

void compute_with_acc_wrapper(float*a, float*b, float*c, size_t count) {
  compute(a,b,c,count);
}

void test_mul_from_external_lib(float*a, float*b, float*c, size_t count) {
  compute(a,b,c,count, true);
}